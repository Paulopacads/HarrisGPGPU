#include "hip/hip_runtime.h"
#include "harris.hh"
#include "convolve.hh"
#include "morph.hh"
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include <chrono>
#include <iostream>

__global__ void set_array_bool(float *mat, int mat_rows, int mat_cols, bool value) {
        
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    mat[i * mat_cols + j] = value;
}

__global__ void set_bool_inferior(bool* output, float *mat, int mat_rows, int mat_cols, float threshold) {
        
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    output[i * mat_cols + j] = mat[i * mat_cols + j] > threshold;
}

__global__ void set_bool_equal(bool* output, float *mat, int mat_rows, int mat_cols, float threshold) {
        
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    output[i * mat_cols + j] = mat[i * mat_cols + j] == threshold;
}

__global__ void set_bool_inverse(bool* m1, bool *m2, int mat_rows, int mat_cols) {
        
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    m1[i * mat_cols + j] = !(m1[i * mat_cols + j] && m2[i * mat_cols + j]);
}

matrix<float> *compute_harris_response(matrix<uint8_t> *img) {
  int derivativeKernelSize = 1;
  int opening_size = 1;

  tuple_matrix<float> tupleImxy =
      gauss_derivatives(img, derivativeKernelSize);

  matrix<float> *gauss = gauss_kernel(opening_size);

  matrix<float> *imxx =
      mat_multiply_element_wise(tupleImxy.mat1, tupleImxy.mat1);
  matrix<float> *imyy =
      mat_multiply_element_wise(tupleImxy.mat2, tupleImxy.mat2);
  matrix<float> *imxy =
      mat_multiply_element_wise(tupleImxy.mat1, tupleImxy.mat2);

  matrix<float> *wxx = convolve(imxx, gauss);
  matrix<float> *wxy = convolve(imxy, gauss);
  matrix<float> *wyy = convolve(imyy, gauss);

  matrix<float> *wxxwyy = mat_multiply_element_wise(wxx, wyy);
  matrix<float> *wxyxy = mat_multiply_element_wise(wxy, wxy);

  matrix<float> *wdet = mat_diff_element_wise(wxxwyy, wxyxy);

  matrix<float> *wtr = mat_add_element_wise(wxx, wyy);

  matrix<float> *wtr1 =  *wtr + 1;
  
  matrix<float> *res = mat_divide_element_wise(wdet, wtr1);

  hipFree(tupleImxy.mat1->values);
  hipFree(tupleImxy.mat2->values);
  delete gauss;
  delete imxx;
  delete imyy;
  delete imxy;
  hipFree(wxx->values);
  hipFree(wyy->values);
  hipFree(wxy->values);
  delete wxxwyy;
  delete wxyxy;
  delete wdet;
  delete wtr;
  delete wtr1;

  return res;
}


matrix<int> *detect_harris_points(matrix<uint8_t> *image_gray, int max_keypoints,
                                     int min_distance, float threshold)
{
    // 1. Compute Harris corner response
    auto time1 = std::chrono::system_clock::now();

    matrix<float> *harris_resp = compute_harris_response(image_gray);

    auto time2 = std::chrono::system_clock::now();
    std::chrono::duration<double> diff = time2 - time1;
    std::cout << "Compute Harris corner response: " << diff.count() << "s" << std::endl;

    // 2. Filtering
    // 2.0 Mask init: all our filtering is performed using a mask
    matrix<bool> *detect_mask = new matrix<bool>(harris_resp->rows, harris_resp->cols);
    for (int i = 0; i < detect_mask->rows * detect_mask->cols; i++) {
        (*detect_mask)[i] = true;
    }

    time1 = std::chrono::system_clock::now();
    diff = time1 - time2;
    std::cout << "Filtering: " << diff.count() << "s" << std::endl;

    // 2.2 Response threshold
    uint8_t min_harris_resp = harris_resp->min();
    auto new_tresh = min_harris_resp + threshold * (harris_resp->max() - min_harris_resp);

    // remove low response elements
    matrix<bool> *mask_harris = new matrix<bool>(harris_resp->rows, harris_resp->cols);
    for (int i = 0; i < mask_harris->rows * mask_harris->cols; ++i) {
        (*mask_harris)[i] = (*harris_resp)[i] > new_tresh ? true : false;
    }

    for (int i = 0; i < detect_mask->rows * detect_mask->cols; i++) {
        if (!((*detect_mask)[i] && (*mask_harris)[i]))
            (*detect_mask)[i] = false;
    }

    time2 = std::chrono::system_clock::now();
    diff = time2 - time1;
    std::cout << "Response threshold: " << diff.count() << "s" << std::endl;

    // 2.3 Non-maximal suppression
    // dil is an image where each local maxima value is propagated to its neighborhood (display it!)
    matrix<bool> *kernel = getStructuringElement(min_distance, min_distance);
    matrix<float> *dil = dilate(harris_resp, kernel);

    // we want to keep only elements which are local maximas in their neighborhood
    matrix<bool> *harris_resp_dil = new matrix<bool>(harris_resp->rows, harris_resp->cols);
    for (int i = 0; i < harris_resp->rows * harris_resp->cols; ++i) {
        (*harris_resp_dil)[i] = (*harris_resp)[i] == (*dil)[i] ? true : false; // keep only local maximas by comparing dil and harris_resp
    }
    for (int i = 0; i < harris_resp_dil->rows * harris_resp_dil->cols; ++i) {
        if (!((*detect_mask)[i] && (*harris_resp_dil)[i]))
            (*detect_mask)[i] = false;
    }

    time1 = std::chrono::system_clock::now();
    diff = time1 - time2;
    std::cout << "Non-maximal suppression: " << diff.count() << "s" << std::endl;

    // 3. Select, sort and filter candidates

    // get coordinates of candidates
    matrix<int> *candidates_coords = detect_mask->non_zero_transposed();

    int nb_candidates = candidates_coords->rows;
    
    // ...and their values
    matrix<float> *candidates_values = new matrix<float>(1, nb_candidates);
    for (int i = 0, j = 0; i < harris_resp->rows * harris_resp->cols; ++i) {
        if ((*detect_mask)[i])
            (*candidates_values)[j++] = (*harris_resp)[i];
    }

    // sort candidates
    matrix<int> *sorted_indices = new matrix<int>(1, nb_candidates);
    for (int i = 0; i < sorted_indices->rows * sorted_indices->cols; ++i) {
        (*sorted_indices)[i] = i;
    }

    //////// TEST ////////

    matrix<int> *test_indices = new matrix<int>(1, 5);
    for (int i = 0; i < test_indices->rows * test_indices->cols; ++i) {
        (*test_indices)[i] = i;
    }

    matrix<float> *test_values = new matrix<float>(1, 5);
    (*test_values)[0] = 0.3;
    (*test_values)[1] = 0.1;
    (*test_values)[2] = 0.2;
    (*test_values)[3] = 0.5;
    (*test_values)[4] = 0.4;

    printf("////// TEST //////\nbefore sort:\n");
    for (int i = 0; i < 5; ++i) {
        printf("value: %f , indice: %d", (*test_values)[i], (*test_indices)[i]);
    }
    thrust::sort_by_key(thrust::host, test_values, test_values + 5, test_indices);
    printf("////// TEST //////\nafter sort:\n");
    for (int i = 0; i < 5; ++i) {
        printf("value: %f , indice: %d", (*test_values)[i], (*test_indices)[i]);
    }

    //////// TEST ///////
    bubbleSort(sorted_indices, candidates_values, nb_candidates);

    // keep only the bests
    if (max_keypoints > nb_candidates)
        max_keypoints = nb_candidates;

    matrix<int> *best_corners_coordinates = new matrix<int>(max_keypoints, 2);
    for (int i = 0; i < max_keypoints; ++i) {
        (*best_corners_coordinates)[i * 2] = (*candidates_coords)[(*sorted_indices)[i] * 2];
        (*best_corners_coordinates)[i * 2 + 1] = (*candidates_coords)[(*sorted_indices)[i] * 2 + 1];
    }

    time2 = std::chrono::system_clock::now();
    diff = time2 - time1;
    std::cout << "Select, sort and filter candidates: " << diff.count() << "s" << std::endl;

    delete harris_resp;
    delete detect_mask;
    delete mask_harris;
    hipFree(kernel->values);
    hipFree(dil->values);
    delete harris_resp_dil;
    delete candidates_coords;
    delete candidates_values;
    delete sorted_indices;

    return best_corners_coordinates;
}