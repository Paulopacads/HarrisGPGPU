#include "hip/hip_runtime.h"
#include "harris.hh"
#include "convolve.hh"
#include "morph.hh"

#include <chrono>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void set_array_bool(bool *mat, int mat_rows, int mat_cols, bool value) {
        
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    mat[i * mat_cols + j] = value;
}

__global__ void set_bool_inferior(bool* output, float *mat, int mat_rows, int mat_cols, float threshold) {
        
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    output[i * mat_cols + j] = mat[i * mat_cols + j] > threshold;
}

__global__ void set_bool_equal(bool*output, float* mat1, float*mat2, int mat_rows, int mat_cols) {
        
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    output[i * mat_cols + j] = mat1[i * mat_cols + j] == mat1[i * mat_cols + j];
}

__global__ void set_bool_inverse(bool* m1, bool *m2, int mat_rows, int mat_cols) {
        
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (!(m1[i * mat_cols + j] && m2[i * mat_cols + j]))
        m1[i * mat_cols + j] = false;
}

matrix<bool>* create_array_bool(int mat_rows, int mat_cols, bool value) {
    bool* output;

    hipMallocManaged(&output, mat_rows * mat_cols * sizeof(bool));
    gpuErrchk(hipGetLastError());

    matrix<bool> *detect_mask = new matrix<bool>(mat_rows, mat_cols, output);
    
    int tx = 24;
    int ty = 16;

    dim3 blocks(mat_cols / tx, mat_rows / ty);
    dim3 threads(tx, ty);
    
    set_array_bool<<<blocks, threads>>>(output, mat_rows, mat_cols, true);
    gpuErrchk(hipGetLastError());


    return detect_mask;
}

matrix<bool> *create_mask_harris(int mat_rows, int mat_cols, float threshold, float* harris_resp) {
    bool* output;

    hipMalloc(&output, mat_rows * mat_cols * sizeof(bool));
    gpuErrchk(hipGetLastError());

    matrix<bool> *mask = new matrix<bool>(mat_rows, mat_cols, output);
    
    int tx = 24;
    int ty = 16;

    dim3 blocks(mat_cols / tx, mat_rows / ty);
    dim3 threads(tx, ty);
    
    set_bool_inferior<<<blocks, threads>>>(output, harris_resp, mat_rows, mat_cols, threshold);
    gpuErrchk(hipGetLastError());

    return mask;
}

void matrix_compare_inverse(bool* m1, bool* m2, int mat_rows, int mat_cols) {
    int tx = 24;
    int ty = 16;

    dim3 blocks(mat_cols / tx, mat_rows / ty);
    dim3 threads(tx, ty);

    set_bool_inverse<<<blocks, threads>>>(m1, m2, mat_rows, mat_cols);
    gpuErrchk(hipGetLastError());
}

matrix<bool>* matrix_compare_equal(float* m1, float* m2, int mat_rows, int mat_cols) {
    bool* output;

    hipMalloc(&output, mat_rows * mat_cols * sizeof(bool));
    gpuErrchk(hipGetLastError());

    matrix<bool> *mask = new matrix<bool>(mat_rows, mat_cols, output);
    
    int tx = 24;
    int ty = 16;

    dim3 blocks(mat_cols / tx, mat_rows / ty);
    dim3 threads(tx, ty);

    set_bool_equal<<<blocks, threads>>>(output, m1, m2, mat_rows, mat_cols);
    gpuErrchk(hipGetLastError());

    return mask;
}

matrix<float> *compute_harris_response(matrix<uint8_t> *img) {
  int derivativeKernelSize = 1;
  int opening_size = 1;

  tuple_matrix<float> tupleImxy =
      gauss_derivatives(img, derivativeKernelSize);

  matrix<float> *gauss = gauss_kernel(opening_size);

  matrix<float> *imxx =
      mat_multiply_element_wise(tupleImxy.mat1, tupleImxy.mat1);
  matrix<float> *imyy =
      mat_multiply_element_wise(tupleImxy.mat2, tupleImxy.mat2);
  matrix<float> *imxy =
      mat_multiply_element_wise(tupleImxy.mat1, tupleImxy.mat2);

  float *gauss_gpu;
  hipMalloc((void **) &gauss_gpu, gauss->rows * gauss->cols * sizeof(float));

  hipMemcpy(gauss_gpu, gauss->values, gauss->rows * gauss->cols * sizeof(float), hipMemcpyHostToDevice);
  gpuErrchk(hipGetLastError());

  matrix<float> *wxx = convolve(imxx, gauss_gpu, gauss->rows, gauss->cols);
  matrix<float> *wxy = convolve(imxy, gauss_gpu, gauss->rows, gauss->cols);
  matrix<float> *wyy = convolve(imyy, gauss_gpu, gauss->rows, gauss->cols);

  matrix<float> *wxxwyy = mat_multiply_element_wise(wxx, wyy);
  matrix<float> *wxyxy = mat_multiply_element_wise(wxy, wxy);

  matrix<float> *wdet = mat_diff_element_wise(wxxwyy, wxyxy);

  matrix<float> *wtr = mat_add_element_wise(wxx, wyy);

  matrix<float> *wtr1 =  *wtr + 1;
  
  matrix<float> *res = mat_divide_element_wise(wdet, wtr1);

  hipFree(tupleImxy.mat1->values);
  hipFree(tupleImxy.mat2->values);
  delete gauss;
  hipFree(gauss_gpu);
  hipFree(imxx->values);
  hipFree(imyy->values);
  hipFree(imxy->values);
  hipFree(wxx->values);
  hipFree(wyy->values);
  hipFree(wxy->values);
  hipFree(wxxwyy->values);
  hipFree(wxyxy->values);
  delete wdet;
  delete wtr;
  delete wtr1;

  return res;
}

matrix<int> *detect_harris_points(matrix<uint8_t> *image_gray, int max_keypoints,
                                     int min_distance, float threshold)
{
    // 1. Compute Harris corner response
    auto time1 = std::chrono::system_clock::now();

    matrix<float> *harris_resp = compute_harris_response(image_gray);

    auto time2 = std::chrono::system_clock::now();
    std::chrono::duration<double> diff = time2 - time1;
    std::cout << "Compute Harris corner response: " << diff.count() << "s" << std::endl;

    float* harris_resp_cu; 
    hipMalloc(&harris_resp_cu, harris_resp->cols * harris_resp -> rows * sizeof(float));
    gpuErrchk(hipGetLastError());

    hipMemcpy(harris_resp_cu, harris_resp->values, harris_resp->cols * harris_resp -> rows * sizeof(float), hipMemcpyHostToDevice);
    gpuErrchk(hipGetLastError());
    // 2. Filtering
    // 2.0 Mask init: all our filtering is performed using a mask
    matrix<bool> *detect_mask = create_array_bool(harris_resp->rows, harris_resp->cols, true);

    
    time1 = std::chrono::system_clock::now();
    diff = time1 - time2;
    std::cout << "Filtering: " << diff.count() << "s" << std::endl;

    // 2.2 Response threshold
    float min_harris_resp = harris_resp->min();
    float new_tresh = min_harris_resp + threshold * (harris_resp->max() - min_harris_resp);

    // remove low response elements
    matrix<bool> *mask_harris = create_mask_harris(harris_resp->rows, harris_resp->cols, new_tresh, harris_resp_cu);    
    matrix_compare_inverse(detect_mask->values, mask_harris->values, harris_resp->rows, harris_resp->cols);

    
    time2 = std::chrono::system_clock::now();
    diff = time2 - time1;
    std::cout << "Response threshold: " << diff.count() << "s" << std::endl;

    // 2.3 Non-maximal suppression
    // dil is an image where each local maxima value is propagated to its neighborhood (display it!)
    matrix<bool> *kernel = getStructuringElement(min_distance, min_distance);
    matrix<float> *dil = dilate(harris_resp, kernel);

    // we want to keep only elements which are local maximas in their neighborhood
    matrix<bool> *harris_resp_dil = matrix_compare_equal(dil->values, harris_resp_cu, harris_resp->rows, harris_resp->cols);
    matrix_compare_inverse(detect_mask->values, harris_resp_dil->values, harris_resp->rows, harris_resp->cols);
    gpuErrchk(hipDeviceSynchronize());

    time1 = std::chrono::system_clock::now();
    diff = time1 - time2;
    std::cout << "Non-maximal suppression: " << diff.count() << "s" << std::endl;

    // 3. Select, sort and filter candidates

    // get coordinates of candidates
    matrix<int> *candidates_coords = detect_mask->non_zero_transposed();

    int nb_candidates = candidates_coords->rows;
    
    // ...and their values
    matrix<float> *candidates_values = new matrix<float>(1, nb_candidates);
    for (int i = 0, j = 0; i < harris_resp->rows * harris_resp->cols; ++i) {
        if ((*detect_mask)[i])
            (*candidates_values)[j++] = (*harris_resp)[i];
    }

    // sort candidates
    int *sorted_indices = (int *) malloc(nb_candidates * sizeof(int));
    for (int i = 0; i < nb_candidates; ++i) {
        sorted_indices[i] = i;
    }

    float *test_values = (float *) malloc(nb_candidates * sizeof(float));
    for (int i = 0; i < nb_candidates; ++i) {
        test_values[i] = (*candidates_values)[i];
    }

    thrust::sort_by_key(thrust::host, test_values, test_values + nb_candidates, sorted_indices);
    // keep only the bests

    if (max_keypoints > nb_candidates)
        max_keypoints = nb_candidates;

    matrix<int> *best_corners_coordinates = new matrix<int>(max_keypoints, 2);
    for (int i = 0; i < max_keypoints; ++i) {
        (*best_corners_coordinates)[i * 2] = (*candidates_coords)[sorted_indices[i] * 2];
        (*best_corners_coordinates)[i * 2 + 1] = (*candidates_coords)[sorted_indices[i] * 2 + 1];
    }

    time2 = std::chrono::system_clock::now();
    diff = time2 - time1;
    std::cout << "Select, sort and filter candidates: " << diff.count() << "s" << std::endl;

    delete harris_resp;
    hipFree(harris_resp_cu);
    hipFree(detect_mask->values);
    hipFree(mask_harris->values);
    hipFree(kernel->values);
    hipFree(dil->values);
    hipFree(harris_resp_dil->values);
    delete candidates_coords;
    delete candidates_values;
    free(sorted_indices);

    return best_corners_coordinates;
}