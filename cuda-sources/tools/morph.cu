#include "hip/hip_runtime.h"
#include "morph.hh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void dilate_pixel(float *output, float *m1, bool *m2, int m1_rows, int m2_rows,
    int m1_cols, int m2_cols) {
        
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    int x_pad = m2_rows / 2;
    int y_pad = m2_cols / 2;

    float max = 0;

    for (int k = 0; k < m2_rows; k++) {
        for (int l = 0; l < m2_cols; l++) {
            if (m2[k * m2_cols + l] && i - x_pad + k >= 0
            && i - x_pad + k < m1_rows && j - y_pad + l >= 0
            && j - y_pad + l < m1_cols) {
                float tmp = m1[(i - x_pad + k) * m1_cols + j - y_pad + l];
                if (tmp > max)
                    max = tmp;
            }
        }
    }
    output[i * m1_cols + j] = max;
}

matrix<float> *dilate(matrix<float> *m1, matrix<bool> *m2) {
    int tx = 24;
    int ty = 16;

    dim3 blocks(m1->cols / tx, m1->rows / ty);
    dim3 threads(tx, ty);

    float *output_gpu;
    float *m1_gpu;
    bool *m2_gpu;

    hipMallocManaged(&output_gpu,  m1->rows * m1->cols * sizeof(float));
    gpuErrchk(hipGetLastError());

    hipMalloc((void **) &m1_gpu, m1->rows * m1->cols * sizeof(float));
    hipMalloc((void **) &m2_gpu, m2->rows * m2->cols * sizeof(bool));

    matrix<float> *output = new matrix<float>(m1->rows, m1->cols, output_gpu);

    hipMemcpy(m1_gpu, m1->values, m1->rows * m1->cols * sizeof(float), hipMemcpyHostToDevice);
    gpuErrchk(hipGetLastError());

    hipMemcpy(m2_gpu, m2->values, m2->rows * m2->cols * sizeof(bool), hipMemcpyHostToDevice);
    gpuErrchk(hipGetLastError());
    
    dilate_pixel<<<blocks, threads>>>(output->values, m1_gpu, m2_gpu,
    m1->rows, m2->rows, m1->cols, m2->cols);
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipFree(m1_gpu);
    hipFree(m2_gpu);

    return output;
}

matrix<bool> *getStructuringElement(int rows, int cols)
{
    int i, j;
    int r = 0, c = 0;
    double inv_r2 = 0;

    r = rows/2;
    c = cols/2;
    inv_r2 = r ? 1./((double)r*r) : 0;

    matrix<bool> *elem = new matrix<bool>(rows, cols);

    for( i = 0; i < rows; i++ )
    {
        int j1 = 0, j2 = 0;
        int dy = i - r;
        if( std::abs(dy) <= r )
        {
            int dx = c*std::sqrt((r*r - dy*dy)*inv_r2);
            j1 = std::max( c - dx, 0 );
            j2 = std::min( c + dx + 1, cols );
        }

        for( j = 0; j < j1; j++ )
            (*elem)[i * cols + j] = false;
        for( ; j < j2; j++ )
            (*elem)[i * cols + j] = true;
        for( ; j < rows; j++ )
            (*elem)[i * cols + j] = false;
    }

    return elem;
}
