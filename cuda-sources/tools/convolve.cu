#include "hip/hip_runtime.h"
#include "convolve.hh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template <typename num>
__global__ void convolve_pixel(float *output, num *m1, float *m2, int m1_rows, int m2_rows,
    int m1_cols, int m2_cols) {

    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    int x_pad = m2_rows / 2;
    int y_pad = m2_cols / 2;

    float conv = 0;

    for (int k = 0; k < m2_rows; k++) {
        for (int l = 0; l < m2_cols; l++) {
            float value = 0;
            if (m2[k * m2_cols + l] && i - x_pad + k >= 0
            && i - x_pad + k < m1_rows && j - y_pad + l >= 0
            && j - y_pad + l < m1_cols) {
                value = m1[(i - x_pad + k) * m1_cols + j - y_pad + l];
            }
            conv += m2[k * m2_cols + l] * value;
        }
    }
    output[i * m1_cols + j] = conv;
}

matrix<float> *convolve(uint8_t *m1, matrix<float> *m2, int m1_rows, int m1_cols) {
    int tx = 24;
    int ty = 16;

    dim3 blocks(m1_cols / tx, m1_rows / ty);
    dim3 threads(tx, ty);

    float *output_gpu;
    float *m2_gpu;

    hipMallocManaged(&output_gpu,  m1_rows * m1_cols * sizeof(float));
    gpuErrchk(hipGetLastError());

    hipMalloc((void **) &m2_gpu, m2->rows * m2->cols * sizeof(float));

    matrix<float> *output = new matrix<float>(m1_rows, m1_cols, output_gpu);

    hipMemcpy(m2_gpu, m2->values, m2->rows * m2->cols * sizeof(float), hipMemcpyHostToDevice);
    gpuErrchk(hipGetLastError());
    
    convolve_pixel<<<blocks, threads>>>(output->values, m1, m2_gpu,
    m1_rows, m2->rows, m1_cols, m2->cols);
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipFree(m2_gpu);

    return output;
}

matrix<float> *convolve(matrix<float> *m1, float *m2, int m2_rows, int m2_cols) {
    int tx = 24;
    int ty = 16;

    dim3 blocks(m1->cols / tx, m1->rows / ty);
    dim3 threads(tx, ty);

    float *output_gpu;
    float *m1_gpu;

    hipMallocManaged(&output_gpu,  m1->rows * m1->cols * sizeof(float));
    gpuErrchk(hipGetLastError());

    hipMalloc((void **) &m1_gpu, m1->rows * m1->cols * sizeof(float));

    matrix<float> *output = new matrix<float>(m1->rows, m1->cols, output_gpu);

    hipMemcpy(m1_gpu, m1->values, m1->rows * m1->cols * sizeof(float), hipMemcpyHostToDevice);
    gpuErrchk(hipGetLastError());
    
    convolve_pixel<<<blocks, threads>>>(output->values, m1_gpu, m2,
    m1->rows, m2_rows, m1->cols, m2_cols);
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipFree(m1_gpu);

    return output;
}

template void __global__ convolve_pixel<uint8_t>(float *output, uint8_t *m1, float *m2, int m1_rows, int m2_rows, int m1_cols, int m2_cols);
template void __global__ convolve_pixel<float>(float *output, float *m1, float *m2, int m1_rows, int m2_rows, int m1_cols, int m2_cols);